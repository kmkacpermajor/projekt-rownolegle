#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <unordered_map>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "config.h"

#define MD5_DIGEST_LENGTH 16
#define SHA256_DIGEST_LENGTH 32

#define SHA256_BLOCK_SIZE 32            // SHA256 outputs a 32 byte digest

typedef struct {
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} CUDA_SHA256_CTX;

#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

__constant__ WORD k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__device__  __forceinline__ void cuda_sha256_transform(CUDA_SHA256_CTX *ctx, const BYTE data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for ( ; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

	for (i = 0; i < 64; ++i) {
		t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
		t2 = EP0(a) + MAJ(a,b,c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

__device__ void cuda_sha256_init(CUDA_SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void cuda_sha256_update(CUDA_SHA256_CTX *ctx, const BYTE data[], size_t len)
{
	WORD i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_sha256_final(CUDA_SHA256_CTX *ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	cuda_sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}

__device__ void kernel_sha256_hash(BYTE* indata, WORD inlen, BYTE* outdata)
{
	CUDA_SHA256_CTX ctx;
	cuda_sha256_init(&ctx);
	cuda_sha256_update(&ctx, indata, inlen);
	cuda_sha256_final(&ctx, outdata);
}

#define MD5_BLOCK_SIZE 16               // MD5 outputs a 16 byte digest

typedef struct {
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[4];
} CUDA_MD5_CTX;

#define F(x,y,z) ((x & y) | (~x & z))
#define G(x,y,z) ((x & z) | (y & ~z))
#define H(x,y,z) (x ^ y ^ z)
#define I(x,y,z) (y ^ (x | ~z))

#define FF(a,b,c,d,m,s,t) { a += F(b,c,d) + m + t; \
                            a = b + ROTLEFT(a,s); }
#define GG(a,b,c,d,m,s,t) { a += G(b,c,d) + m + t; \
                            a = b + ROTLEFT(a,s); }
#define HH(a,b,c,d,m,s,t) { a += H(b,c,d) + m + t; \
                            a = b + ROTLEFT(a,s); }
#define II(a,b,c,d,m,s,t) { a += I(b,c,d) + m + t; \
                            a = b + ROTLEFT(a,s); }

/*********************** FUNCTION DEFINITIONS ***********************/
__device__ void cuda_md5_transform(CUDA_MD5_CTX *ctx, const BYTE data[])
{
	WORD a, b, c, d, m[16], i, j;

	// MD5 specifies big endian byte order, but this implementation assumes a little
	// endian byte order CPU. Reverse all the bytes upon input, and re-reverse them
	// on output (in md5_final()).
	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j]) + (data[j + 1] << 8) + (data[j + 2] << 16) + (data[j + 3] << 24);

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];

	FF(a,b,c,d,m[0],  7,0xd76aa478);
	FF(d,a,b,c,m[1], 12,0xe8c7b756);
	FF(c,d,a,b,m[2], 17,0x242070db);
	FF(b,c,d,a,m[3], 22,0xc1bdceee);
	FF(a,b,c,d,m[4],  7,0xf57c0faf);
	FF(d,a,b,c,m[5], 12,0x4787c62a);
	FF(c,d,a,b,m[6], 17,0xa8304613);
	FF(b,c,d,a,m[7], 22,0xfd469501);
	FF(a,b,c,d,m[8],  7,0x698098d8);
	FF(d,a,b,c,m[9], 12,0x8b44f7af);
	FF(c,d,a,b,m[10],17,0xffff5bb1);
	FF(b,c,d,a,m[11],22,0x895cd7be);
	FF(a,b,c,d,m[12], 7,0x6b901122);
	FF(d,a,b,c,m[13],12,0xfd987193);
	FF(c,d,a,b,m[14],17,0xa679438e);
	FF(b,c,d,a,m[15],22,0x49b40821);

	GG(a,b,c,d,m[1],  5,0xf61e2562);
	GG(d,a,b,c,m[6],  9,0xc040b340);
	GG(c,d,a,b,m[11],14,0x265e5a51);
	GG(b,c,d,a,m[0], 20,0xe9b6c7aa);
	GG(a,b,c,d,m[5],  5,0xd62f105d);
	GG(d,a,b,c,m[10], 9,0x02441453);
	GG(c,d,a,b,m[15],14,0xd8a1e681);
	GG(b,c,d,a,m[4], 20,0xe7d3fbc8);
	GG(a,b,c,d,m[9],  5,0x21e1cde6);
	GG(d,a,b,c,m[14], 9,0xc33707d6);
	GG(c,d,a,b,m[3], 14,0xf4d50d87);
	GG(b,c,d,a,m[8], 20,0x455a14ed);
	GG(a,b,c,d,m[13], 5,0xa9e3e905);
	GG(d,a,b,c,m[2],  9,0xfcefa3f8);
	GG(c,d,a,b,m[7], 14,0x676f02d9);
	GG(b,c,d,a,m[12],20,0x8d2a4c8a);

	HH(a,b,c,d,m[5],  4,0xfffa3942);
	HH(d,a,b,c,m[8], 11,0x8771f681);
	HH(c,d,a,b,m[11],16,0x6d9d6122);
	HH(b,c,d,a,m[14],23,0xfde5380c);
	HH(a,b,c,d,m[1],  4,0xa4beea44);
	HH(d,a,b,c,m[4], 11,0x4bdecfa9);
	HH(c,d,a,b,m[7], 16,0xf6bb4b60);
	HH(b,c,d,a,m[10],23,0xbebfbc70);
	HH(a,b,c,d,m[13], 4,0x289b7ec6);
	HH(d,a,b,c,m[0], 11,0xeaa127fa);
	HH(c,d,a,b,m[3], 16,0xd4ef3085);
	HH(b,c,d,a,m[6], 23,0x04881d05);
	HH(a,b,c,d,m[9],  4,0xd9d4d039);
	HH(d,a,b,c,m[12],11,0xe6db99e5);
	HH(c,d,a,b,m[15],16,0x1fa27cf8);
	HH(b,c,d,a,m[2], 23,0xc4ac5665);

	II(a,b,c,d,m[0],  6,0xf4292244);
	II(d,a,b,c,m[7], 10,0x432aff97);
	II(c,d,a,b,m[14],15,0xab9423a7);
	II(b,c,d,a,m[5], 21,0xfc93a039);
	II(a,b,c,d,m[12], 6,0x655b59c3);
	II(d,a,b,c,m[3], 10,0x8f0ccc92);
	II(c,d,a,b,m[10],15,0xffeff47d);
	II(b,c,d,a,m[1], 21,0x85845dd1);
	II(a,b,c,d,m[8],  6,0x6fa87e4f);
	II(d,a,b,c,m[15],10,0xfe2ce6e0);
	II(c,d,a,b,m[6], 15,0xa3014314);
	II(b,c,d,a,m[13],21,0x4e0811a1);
	II(a,b,c,d,m[4],  6,0xf7537e82);
	II(d,a,b,c,m[11],10,0xbd3af235);
	II(c,d,a,b,m[2], 15,0x2ad7d2bb);
	II(b,c,d,a,m[9], 21,0xeb86d391);

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
}

__device__ void cuda_md5_init(CUDA_MD5_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
}

__device__ void cuda_md5_update(CUDA_MD5_CTX *ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_md5_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_md5_final(CUDA_MD5_CTX *ctx, BYTE hash[])
{
	size_t i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else if (ctx->datalen >= 56) {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_md5_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[56] = ctx->bitlen;
	ctx->data[57] = ctx->bitlen >> 8;
	ctx->data[58] = ctx->bitlen >> 16;
	ctx->data[59] = ctx->bitlen >> 24;
	ctx->data[60] = ctx->bitlen >> 32;
	ctx->data[61] = ctx->bitlen >> 40;
	ctx->data[62] = ctx->bitlen >> 48;
	ctx->data[63] = ctx->bitlen >> 56;
	cuda_md5_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and MD uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (i * 8)) & 0x000000ff;
	}
}

__device__ void kernel_md5_hash(BYTE* indata, WORD inlen, BYTE* outdata)
{
	CUDA_MD5_CTX ctx;
	cuda_md5_init(&ctx);
	cuda_md5_update(&ctx, indata, inlen);
	cuda_md5_final(&ctx, outdata);
}

__device__ int my_strlen(const char* str) {
    int len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

__device__ void my_strncpy(char* dest, const char* src, int n) {
    for (int i = 0; i < n; i++) {
        if (src[i] != '\0') {
            dest[i] = src[i];
        } else {
            dest[i] = '\0';
        }
    }
}

__device__ void my_sprintf(char* str, const char* format, unsigned int value) {
    // Assuming format is always "%02x" and str has enough space
    const char hex_chars[] = "0123456789abcdef";
    str[0] = hex_chars[(value >> 4) & 0xF];
    str[1] = hex_chars[value & 0xF];
    str[2] = '\0'; // Null-terminate
}

__device__ int my_strncmp(const char* s1, const char* s2, int n) {
    for (int i = 0; i < n; i++) {
        if (s1[i] != s2[i]) {
            return s1[i] - s2[i];
        }
        if (s1[i] == '\0') {
            return 0;
        }
    }
    return 0;
}

// eventem albo zmienną atomową
// otagowac zmienna zeby nie wrzucić do rejestru
// przekompilować do ptx
// stałe w definie
__global__ void crackHashes(const char* d_hashes, const char* d_dictionary, int dict_size, int hash_length, char* d_results, int hash_type) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dict_size) {
        char word[100] = { 0 };
        my_strncpy(word, &d_dictionary[idx * 100], 100);
        int word_len = my_strlen(word);
        BYTE hash[32];

        if (hash_type == 0) {
            kernel_md5_hash((BYTE*)word, word_len, hash);
        } else {
            kernel_sha256_hash((BYTE*)word, word_len, hash);
        }

        
        char computed_hash[64];
        for (int j = 0; j < hash_length; j++) {
            my_sprintf(&computed_hash[j * 2], "%02x", hash[j]);
        }

        if (my_strncmp(computed_hash, &d_hashes[idx * 64], 64) == 0) {
            my_strncpy(&d_results[idx * 100], word, 100);
        } else {
            d_results[idx * 100] = '\0';
        }
    }
}

std::string extractHash(const std::string& input) {
    size_t firstDollarPos = input.find('$');
    if (firstDollarPos != std::string::npos) {
        size_t secondDollarPos = input.find('$', firstDollarPos + 1);
        if (secondDollarPos != std::string::npos) {
            size_t thirdDollarPos = input.find('$', secondDollarPos + 1);
            if (thirdDollarPos != std::string::npos) {
                return input.substr(thirdDollarPos + 1);
            }
        }
    }
    return "";
}

int main(int argc, char* argv[]) {
    if (argc != 4 && argc != 6) {
        std::cerr << "Usage: " << argv[0] << " <hash_file> <dictionary_file> <output_file> [<hash_lines> <dict_lines>]\n";
        return 1;
    }

	int login_lines;
	int dict_lines;
	if (argc == 6){
		login_lines = std::atoi(argv[4]);
		dict_lines = std::atoi(argv[5]);
	}

    std::ifstream hashFile(argv[1]);
    if (!hashFile.is_open()) {
        std::cerr << "Error: Couldn't open hash file.\n";
        return 1;
    }

    std::ifstream dictionaryFile(argv[2]);
    if (!dictionaryFile.is_open()) {
        std::cerr << "Error: Couldn't open dictionary file.\n";
        return 1;
    }

    std::ofstream outputFile(argv[3]);
    if (!outputFile.is_open()) {
        std::cerr << "Error: Couldn't create output file.\n";
        return 1;
    }

    std::unordered_map<std::string, std::string> loginToHash;
    std::vector<std::string> dictionary;

    std::string line;
	int i = 0;
    while (std::getline(hashFile, line)) {
        size_t pos = line.find(':');
        if (pos != std::string::npos) {
            std::string login = line.substr(0, pos);
            std::string hash = line.substr(pos + 1);
            loginToHash[login] = hash;
        }
		
		if (argc == 6 && i>login_lines) break;
		i++;
    }

	i = 0;
    while (std::getline(dictionaryFile, line)) {
        dictionary.push_back(line);

		if (argc == 6 && i>dict_lines) break;
		i++;
    }

    int dict_size = dictionary.size();
    char* d_dictionary;
    char* d_hashes;
    char* d_results;
    int hash_length = 32; // SHA256 has 32 bytes output, MD5 has 16 bytes output
    int hash_type = 0; // 0 for MD5, 1 for SHA256

    hipMalloc(&d_dictionary, dict_size * 100 * sizeof(char));
    hipMalloc(&d_hashes, dict_size * 64 * sizeof(char));
    hipMalloc(&d_results, dict_size * 100 * sizeof(char));

    for (const auto& pair : loginToHash) {
        std::string login = pair.first;
        std::string whole_hash = pair.second;
        std::string hash = extractHash(whole_hash);
		std::string salt = whole_hash.substr(whole_hash.find('$', whole_hash.find('$') + 1) + 1, whole_hash.find('$', whole_hash.find('$', whole_hash.find('$') + 1) + 1) - whole_hash.find('$', whole_hash.find('$') + 1) - 1);

        if (whole_hash.find("MD5") != std::string::npos) {
            hash_type = 0;
            hash_length = 16;
        } else if (whole_hash.find("SHA256") != std::string::npos) {
            hash_type = 1;
            hash_length = 32;
        }

        std::vector<char> h_hashes(dict_size * 64, '\0');
        for (int i = 0; i < dict_size; i++) {
            strncpy(&h_hashes[i * 64], hash.c_str(), 64);
        }

		size_t totalSize = dictionary.size() * 100;

		char* h_dictionary = new char[totalSize];
		memset(h_dictionary, ' ', totalSize);

		for (size_t i = 0; i < dictionary.size(); ++i) {
			memcpy(h_dictionary + i * 100, (dictionary[i]+salt).c_str(), 100);
		}

        hipMemcpy(d_dictionary, h_dictionary, dict_size * 100 * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_hashes, h_hashes.data(), dict_size * 64 * sizeof(char), hipMemcpyHostToDevice);

		// tym popróbować
        int blockSize = 256;
        int numBlocks = (dict_size + blockSize - 1) / blockSize;
		// sprawdzic narzut (funkcja cuda nic nie robi)
        crackHashes<<<numBlocks, blockSize>>>(d_hashes, d_dictionary, dict_size, hash_length, d_results, hash_type);
        hipDeviceSynchronize();

        std::vector<char> h_results(dict_size * 100, '\0');
        hipMemcpy(h_results.data(), d_results, dict_size * 100 * sizeof(char), hipMemcpyDeviceToHost);

        for (int i = 0; i < dict_size; i++) {
            if (h_results[i * 100] != '\0') {
				std::string x = &h_results[i * 100];
				x.erase(x.length() - salt.length(), salt.length());
                outputFile << login << ":" << x << "\n";
                break;
            }
        }
    }

    hipFree(d_dictionary);
    hipFree(d_hashes);
    hipFree(d_results);

    hashFile.close();
    dictionaryFile.close();
    outputFile.close();

    return 0;
}
